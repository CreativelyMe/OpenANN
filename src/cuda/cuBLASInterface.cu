#include "cuBLASInterface.cuh"
#include <CUDAcommonUtils.cuh>
#include <cmath>

CUBLASContext CUBLASContext::instance;

CUBLASContext::CUBLASContext()
{
  handleCUBLASError(hipblasCreate(&handle));
}

CUBLASContext::~CUBLASContext()
{
  handleCUBLASError(hipblasDestroy(handle));
}

bool CUBLASContext::allocateMatrix(float** device, int rows, int cols)
{
  const int size = rows*cols * sizeof(float);
  return handleCUDAError(hipMalloc((void**)device, size));
}

bool CUBLASContext::freeMatrix(float* device)
{
  return handleCUDAError(hipFree(device));
}

bool CUBLASContext::setMatrix(const float* host, float* device, int rows, int cols)
{
  if(cols == 1)
  {
    return handleCUBLASError(hipblasSetVector(rows, sizeof(*host), host, 1, device, 1));
  }
  else
  {
    return handleCUBLASError(hipblasSetMatrix(rows, cols, sizeof(*host), host, rows, device, rows));
  }
}

bool CUBLASContext::getMatrix(float* host, const float* device, int rows, int cols)
{
  if(cols == 1)
  {
    return handleCUBLASError(hipblasGetVector(rows, sizeof(*device), device, 1, host, 1));
  }
  else
  {
    return handleCUBLASError(hipblasGetMatrix(rows, cols, sizeof(*device), device, rows, host, rows));
  }
}

bool CUBLASContext::multiplyMatrixMatrix(float* a, float* b, float* result, int rows1, int cols1, int cols2)
{
  float alpha = 1.0, beta = 0.0;
  if(cols2 == 1)
    return handleCUBLASError(hipblasSgemv(handle, HIPBLAS_OP_N, rows1, cols1, &alpha, a, rows1, b, 1, &beta, result, 1));
  else
    return handleCUBLASError(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rows1, cols2, cols1, &alpha, a, rows1, b, cols1, &beta, result, rows1));
}
